
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdint>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

//TODO: adjust kernel parameters
#define BLOCKS_NUM 128
#define THREADS_NUM 128 // must be 2^n

#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

__constant__ hipComplex q1g[4];
__constant__ hipComplex q2g[16];

// utility function necessary to traverse a state
__device__ size_t insert_zero(
  size_t mask,
  size_t offset
)
{
  return ((mask & offset) << 1) | ((~mask) & offset);
}

// allocate an uninitialized state on the device
extern "C"
int32_t get_state (
  hipComplex** state,
  size_t qubits_number
)
{
  size_t size = 1 << qubits_number;
  int32_t status = hipMalloc(state, size * sizeof(hipComplex));
  return status;
}

// copy a state to the host
extern "C"
int32_t copy_to_host (
  hipComplex* state,
  hipComplex* host_state,
  size_t qubits_number
)
{
  size_t size = 1 << qubits_number;
  int32_t status = hipMemcpy(host_state, state, size * sizeof(hipComplex), hipMemcpyDeviceToHost);
  return status;
}

// drop state on the device
extern "C"
int32_t drop_state(
  hipComplex* state
)
{
  int32_t status = hipFree(state);
  return status;
}

// initialize state to the standard one
__global__ void _set2standard (
  hipComplex* state,
  size_t  qubits_number
)
{
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    state[tid] = {0., 0.};
  }
  __syncthreads();
  if ( blockIdx.x == 0 && threadIdx.x == 0 ) {
    state[0] = {1., 0.};
  }
}

extern "C"
void set2standard (
  hipComplex* state,
  size_t  qubits_number
)
{
  _set2standard<<<BLOCKS_NUM, THREADS_NUM>>> (
    state,
    qubits_number
  );
}

// computes l2 norm of a state
__global__ void _norm (
  const hipComplex* state,
  float* result,
  size_t qubits_number
)
{
  __shared__ float cache[THREADS_NUM];
  float tmp = 0;
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    float real = state[tid].x;
    float imag = state[tid].y;
    tmp += real * real + imag * imag;
  }
  cache[threadIdx.x] = tmp;
  __syncthreads();
  int s = blockDim.x / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      cache[threadIdx.x] += cache[threadIdx.x + s];
    }
    __syncthreads();
    s /= 2;
  }
  if (threadIdx.x == 0) {
    result[blockIdx.x] = cache[0];
  }
}

// sets state from host
extern "C"
int32_t set_from_host (
  hipComplex* device_state,
  const hipComplex* host_state,
  size_t qubits_number
)
{
  int32_t memcpy_state = hipMemcpy(device_state, host_state, (1 << qubits_number) * sizeof(hipComplex), hipMemcpyHostToDevice);
  return memcpy_state;
}

extern "C"
int32_t norm (
  const hipComplex* state,
  float* result,
  size_t qubits_number
)
{
  float* device_result;
  float* host_result;
  host_result = (float*)malloc(BLOCKS_NUM * sizeof(float));
  int32_t alloc_status = hipMalloc(&device_result, BLOCKS_NUM * sizeof(float));
  _norm<<<BLOCKS_NUM, THREADS_NUM>>>(state, device_result, qubits_number);
  int32_t memcopy_status = hipMemcpy(
    host_result,
    device_result,
    BLOCKS_NUM * sizeof(float),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i++) {
    *result += host_result[i];
  }
  delete[] host_result;
  int32_t free_status = hipFree(device_result);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// one qubits gate application
__global__ void _q1gate(
  hipComplex* state,
  size_t pos,
  size_t qubits_number
)
{
  size_t mask =  SIZE_MAX << pos;
  size_t stride = 1 << pos;
  size_t size = 1 << qubits_number;
  size_t batch_size = size >> 1;
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(mask, tid);
    hipComplex tmp = hipCaddf(
      hipCmulf(q1g[0], state[btid]),
      hipCmulf(q1g[1], state[btid + stride])
    );
    state[stride + btid] = hipCaddf(
      hipCmulf(q1g[2], state[btid]),
      hipCmulf(q1g[3], state[btid + stride])
    );
    state[btid] = tmp;
  }
}

extern "C"
int32_t q1gate(
  hipComplex* state,
  const hipComplex* gate,
  size_t idx,
  size_t qubits_number
)
{
  int32_t copy_status = hipMemcpyToSymbol(HIP_SYMBOL(q1g), gate, 4 * sizeof(hipComplex));
  _q1gate<<<BLOCKS_NUM, THREADS_NUM>>>(state, idx, qubits_number);
  return copy_status;
}

// two qubits gate application
__global__ void _q2gate(
  hipComplex* state,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  size_t min_pos = MIN(pos1, pos2);
  size_t max_pos = MAX(pos1, pos2);
  size_t min_mask =  SIZE_MAX << min_pos;
  size_t max_mask =  SIZE_MAX << max_pos;
  size_t size = 1 << qubits_number;
  size_t stride1 = 1 << pos1;
  size_t stride2 = 1 << pos2;
  size_t batch_size = size >> 2;
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(max_mask, insert_zero(min_mask, tid));
    hipComplex tmp[4] = { {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.} };
    for (size_t p1 = 0; p1 < 2; p1++) {
      for (size_t p2 = 0; p2 < 2; p2++) {
        for (size_t q1 = 0; q1 < 2; q1++) {
          for (size_t q2 = 0; q2 < 2; q2++) {
            tmp[2 * q2 + q1] = hipCaddf(
              tmp[2 * q2 + q1],
              hipCmulf(
                q2g[8 * q2 + 4 * q1 + 2 * p2 + p1],
                state[stride2 * p2 + stride1 * p1 + btid]
              )
            );
          }
        }
      }
    }
    state[btid] = tmp[0];
    state[btid + stride1] = tmp[1];
    state[btid + stride2] = tmp[2];
    state[btid + stride1 + stride2] = tmp[3];
  }
}

extern "C"
int32_t q2gate(
  hipComplex* state,
  const hipComplex* gate,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  int32_t copy_status = hipMemcpyToSymbol(HIP_SYMBOL(q2g), gate, 16 * sizeof(hipComplex));
  _q2gate<<<BLOCKS_NUM, THREADS_NUM>>>(state, pos2, pos1, qubits_number);
  return copy_status;
}

// one qubit density matrix computation
__global__ void _get_q1density(
  const hipComplex* state,
  hipComplex* density,
  size_t pos,
  size_t qubits_number
)
{
  __shared__ hipComplex cache[4 * THREADS_NUM];
  size_t mask =  SIZE_MAX << pos;
  size_t stride = 1 << pos;
  size_t size = 1 << qubits_number;
  size_t batch_size = size >> 1;
  hipComplex tmp[4] = { 
    {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(mask, tid);
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        tmp[2 * p + q] = hipCaddf(
          tmp[2 * p + q],
          hipCmulf(
            state[p * stride + btid],
            hipConjf(state[q * stride + btid])
          )
        );
      }
    }
  }
  for (int q = 0; q < 2; q++) {
    for (int p = 0; p < 2; p++) {
      cache[2 * p + q + 4 * threadIdx.x] = tmp[2 * p + q];
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q = 0; q < 2; q++) {
        for (int p = 0; p < 2; p++) {
          cache[2 * p + q + 4 * threadIdx.x] = hipCaddf(
            cache[2 * p + q + 4 * threadIdx.x],
            cache[2 * p + q + 4 * (threadIdx.x + s)]
          );
        }
      }
    }
    s /= 2;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        density[2 * p + q + 4 * blockIdx.x] = cache[2 * p + q];
      }
    }
  }
}

extern "C"
int32_t get_q1density(
  const hipComplex* state,
  hipComplex* density,
  size_t pos,
  size_t qubits_number
)
{
  hipComplex* device_density;
  hipComplex* host_density;
  int32_t alloc_status = hipMalloc(&device_density, 4 * BLOCKS_NUM * sizeof(hipComplex));
  _get_q1density<<<BLOCKS_NUM, THREADS_NUM>>>(
    state,
    device_density,
    pos,
    qubits_number
  );
  host_density = (hipComplex*)malloc(4 * BLOCKS_NUM * sizeof(hipComplex));
  int32_t memcopy_status = hipMemcpy(
    host_density,
    device_density,
    4 * BLOCKS_NUM * sizeof(hipComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        density[2 * p + q] = hipCaddf(
          density[2 * p + q],
          host_density[2 * p + q + 4 * i]
        );
      }
    }
  }
  delete[] host_density;
  int32_t free_status = hipFree(device_density);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// two qubit density matrix computation
__global__ void _get_q2density(
  const hipComplex* state,
  hipComplex* density,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  __shared__ hipComplex cache[16 * THREADS_NUM];
  size_t min_pos = MIN(pos1, pos2);
  size_t max_pos = MAX(pos1, pos2);
  size_t min_mask =  SIZE_MAX << min_pos;
  size_t max_mask =  SIZE_MAX << max_pos;
  size_t size = 1 << qubits_number;
  size_t stride1 = 1 << pos1;
  size_t stride2 = 1 << pos2;
  size_t batch_size = size >> 2;
  hipComplex tmp[16] = {
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(max_mask, insert_zero(min_mask, tid));
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            tmp[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              tmp[8 * p2 + 4 * p1 + 2 * q2 + q1],
              hipCmulf(
                state[p2 * stride2 + p1 * stride1 + btid],
                hipConjf(state[q2 * stride2 + q1 * stride1 + btid])
              )
            );
          }
        }
      }
    }
  }
  for (int q1 = 0; q1 < 2; q1++) {
    for (int q2 = 0; q2 < 2; q2++) {
      for (int p1 = 0; p1 < 2; p1++) {
        for (int p2 = 0; p2 < 2; p2++) {
          cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = tmp[8 * p2 + 4 * p1 + 2 * q2 + q1];
        }
      }
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q1 = 0; q1 < 2; q1++) {
        for (int q2 = 0; q2 < 2; q2++) {
          for (int p1 = 0; p1 < 2; p1++) {
            for (int p2 = 0; p2 < 2; p2++) {
              cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = hipCaddf(
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x],
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * (threadIdx.x + s)]
              );
            }
          }
        }
      }
    }
    s /= 2;
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            density[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * blockIdx.x] = cache[8 * p2 + 4 * p1 + 2 * q2 + q1];
          }
        }
      }
    }
  }
}

extern "C"
int32_t get_q2density(
  const hipComplex* state,
  hipComplex* density,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  hipComplex* device_density;
  hipComplex* host_density;
  int32_t alloc_status = hipMalloc(&device_density, 16 * BLOCKS_NUM * sizeof(hipComplex));
  _get_q2density<<<BLOCKS_NUM, THREADS_NUM>>>(
    state,
    device_density,
    pos2,
    pos1,
    qubits_number
  );
  host_density = (hipComplex*)malloc(16 * BLOCKS_NUM * sizeof(hipComplex));
  int32_t memcopy_status = hipMemcpy(
    host_density,
    device_density,
    16 * BLOCKS_NUM * sizeof(hipComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            density[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              density[8 * p2 + 4 * p1 + 2 * q2 + q1],
              host_density[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * i]
            );
          }
        }
      }
    }
  }
  delete[] host_density;
  int32_t free_status = hipFree(device_density);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}
