
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdint>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

//TODO: adjust kernel parameters
#define BLOCKS_NUM 128
#define THREADS_NUM 128 // must be 2^n

#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

__constant__ hipFloatComplex q1g[4];
__constant__ hipFloatComplex q2g[16];

// utility function necessary to traverse a state
__device__ size_t insert_zero(
  size_t mask,
  size_t offset
)
{
  return ((mask & offset) << 1) | ((~mask) & offset);
}

// allocate an uninitialized state on the device
extern "C"
int32_t get_state (
  hipFloatComplex** state,
  size_t qubits_number
)
{
  size_t size = 1 << qubits_number;
  int32_t status = hipMalloc(state, size * sizeof(hipFloatComplex));
  return status;
}

// copy a state to the host
extern "C"
int32_t copy_to_host (
  hipFloatComplex* state,
  hipFloatComplex* host_state,
  size_t qubits_number
)
{
  size_t size = 1 << qubits_number;
  int32_t status = hipMemcpy(host_state, state, size * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
  return status;
}

// drop state on the device
extern "C"
int32_t drop_state(
  hipFloatComplex* state
)
{
  int32_t status = hipFree(state);
  return status;
}

// initialize state to the standard one
__global__ void _set2standard (
  hipFloatComplex* state,
  size_t  qubits_number
)
{
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    state[tid] = {0., 0.};
  }
  __syncthreads();
  if ( blockIdx.x == 0 && threadIdx.x == 0 ) {
    state[0] = {1., 0.};
  }
}

extern "C"
void set2standard (
  hipFloatComplex* state,
  size_t  qubits_number
)
{
  _set2standard<<<BLOCKS_NUM, THREADS_NUM>>> (
    state,
    qubits_number
  );
}

// computes q1 gate gradient from bwd and fwd "states"
__global__ void _q1grad (
  const hipFloatComplex* fwd,
  const hipFloatComplex* bwd,
  hipFloatComplex* grad,
  size_t pos,
  size_t qubits_number
)
{
  __shared__ hipFloatComplex cache[4 * THREADS_NUM];
  size_t mask =  SIZE_MAX << pos;
  size_t stride = 1 << pos;
  size_t size = 1 << qubits_number;
  size_t batch_size = size >> 1;
  hipFloatComplex tmp[4] = { 
    {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(mask, tid);
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        tmp[2 * p + q] = hipCaddf(
          tmp[2 * p + q],
          hipCmulf(
            bwd[p * stride + btid],
            fwd[q * stride + btid]
          )
        );
      }
    }
  }
  for (int q = 0; q < 2; q++) {
    for (int p = 0; p < 2; p++) {
      cache[2 * p + q + 4 * threadIdx.x] = tmp[2 * p + q];
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q = 0; q < 2; q++) {
        for (int p = 0; p < 2; p++) {
          cache[2 * p + q + 4 * threadIdx.x] = hipCaddf(
            cache[2 * p + q + 4 * threadIdx.x],
            cache[2 * p + q + 4 * (threadIdx.x + s)]
          );
        }
      }
    }
    __syncthreads();
    s /= 2;
  }
  if (threadIdx.x == 0) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        grad[2 * p + q + 4 * blockIdx.x] = cache[2 * p + q];
      }
    }
  }
}

extern "C"
int q1grad (
  const hipFloatComplex* fwd,
  const hipFloatComplex* bwd,
  hipFloatComplex* grad,
  size_t pos,
  size_t qubits_number
)
{
  hipFloatComplex* device_grad;
  hipFloatComplex* host_grad;
  int32_t alloc_status = hipMalloc(&device_grad, 4 * BLOCKS_NUM * sizeof(hipFloatComplex));
  _q1grad<<<BLOCKS_NUM, THREADS_NUM>>>(
    fwd,
    bwd,
    device_grad,
    pos,
    qubits_number
  );
  host_grad = (hipFloatComplex*)malloc(4 * BLOCKS_NUM * sizeof(hipFloatComplex));
  int32_t memcopy_status = hipMemcpy(
    host_grad,
    device_grad,
    4 * BLOCKS_NUM * sizeof(hipFloatComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        grad[2 * p + q] = hipCaddf(
          grad[2 * p + q],
          host_grad[2 * p + q + 4 * i]
        );
      }
    }
  }
  delete[] host_grad;
  int32_t free_status = hipFree(device_grad);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// computes q2 hate gradient from fwd and bwd "states"
__global__ void _q2grad (
  const hipFloatComplex* fwd,
  const hipFloatComplex* bwd,
  hipFloatComplex* grad,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  __shared__ hipFloatComplex cache[16 * THREADS_NUM];
  size_t min_pos = MIN(pos1, pos2);
  size_t max_pos = MAX(pos1, pos2);
  size_t min_mask =  SIZE_MAX << min_pos;
  size_t max_mask =  SIZE_MAX << max_pos;
  size_t size = 1 << qubits_number;
  size_t stride1 = 1 << pos1;
  size_t stride2 = 1 << pos2;
  size_t batch_size = size >> 2;
  hipFloatComplex tmp[16] = { 
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(max_mask, insert_zero(min_mask, tid));
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            tmp[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              tmp[8 * p2 + 4 * p1 + 2 * q2 + q1],
              hipCmulf(
                bwd[p2 * stride2 + p1 * stride1 + btid],
                fwd[q2 * stride2 + q1 * stride1 + btid]
              )
            );
          }
        }
      }
    }
  }
  for (int q1 = 0; q1 < 2; q1++) {
    for (int q2 = 0; q2 < 2; q2++) {
      for (int p1 = 0; p1 < 2; p1++) {
        for (int p2 = 0; p2 < 2; p2++) {
          cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = tmp[8 * p2 + 4 * p1 + 2 * q2 + q1];
        }
      }
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q1 = 0; q1 < 2; q1++) {
        for (int q2 = 0; q2 < 2; q2++) {
          for (int p1 = 0; p1 < 2; p1++) {
            for (int p2 = 0; p2 < 2; p2++) {
              cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = hipCaddf(
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x],
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * (threadIdx.x + s)]
              );
            }
          }
        }
      }
    }
    __syncthreads();
    s /= 2;
  }
  if (threadIdx.x == 0) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            grad[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * blockIdx.x] = cache[8 * p2 + 4 * p1 + 2 * q2 + q1];
          }
        }
      }
    }
  }
}

extern "C"
int q2grad (
  const hipFloatComplex* fwd,
  const hipFloatComplex* bwd,
  hipFloatComplex* grad,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  hipFloatComplex* device_grad;
  hipFloatComplex* host_grad;
  int32_t alloc_status = hipMalloc(&device_grad, 16 * BLOCKS_NUM * sizeof(hipFloatComplex));
  _q2grad<<<BLOCKS_NUM, THREADS_NUM>>>(
    fwd,
    bwd,
    device_grad,
    pos2,
    pos1,
    qubits_number
  );
  host_grad = (hipFloatComplex*)malloc(16 * BLOCKS_NUM * sizeof(hipFloatComplex));
  int32_t memcopy_status = hipMemcpy(
    host_grad,
    device_grad,
    16 * BLOCKS_NUM * sizeof(hipFloatComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            grad[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              grad[8 * p2 + 4 * p1 + 2 * q2 + q1],
              host_grad[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * i]
            );
          }
        }
      }
    }
  }
  delete[] host_grad;
  int32_t free_status = hipFree(device_grad);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// sets state from host
extern "C"
int32_t set_from_host (
  hipFloatComplex* device_state,
  const hipFloatComplex* host_state,
  size_t qubits_number
)
{
  int32_t memcpy_state = hipMemcpy(device_state, host_state, (1 << qubits_number) * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
  return memcpy_state;
}

// one qubits gate application
__global__ void _q1gate(
  hipFloatComplex* state,
  size_t pos,
  size_t qubits_number
)
{
  size_t mask =  SIZE_MAX << pos;
  size_t stride = 1 << pos;
  size_t size = 1 << qubits_number;
  size_t batch_size = size >> 1;
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(mask, tid);
    hipFloatComplex tmp = hipCaddf(
      hipCmulf(q1g[0], state[btid]),
      hipCmulf(q1g[1], state[btid + stride])
    );
    state[stride + btid] = hipCaddf(
      hipCmulf(q1g[2], state[btid]),
      hipCmulf(q1g[3], state[btid + stride])
    );
    state[btid] = tmp;
  }
}

extern "C"
int32_t q1gate(
  hipFloatComplex* state,
  const hipFloatComplex* gate,
  size_t idx,
  size_t qubits_number
)
{
  int32_t copy_status = hipMemcpyToSymbol(HIP_SYMBOL(q1g), gate, 4 * sizeof(hipFloatComplex));
  _q1gate<<<BLOCKS_NUM, THREADS_NUM>>>(state, idx, qubits_number);
  return copy_status;
}

// two qubits gate application
__global__ void _q2gate(
  hipFloatComplex* state,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  size_t min_pos = MIN(pos1, pos2);
  size_t max_pos = MAX(pos1, pos2);
  size_t min_mask =  SIZE_MAX << min_pos;
  size_t max_mask =  SIZE_MAX << max_pos;
  size_t size = 1 << qubits_number;
  size_t stride1 = 1 << pos1;
  size_t stride2 = 1 << pos2;
  size_t batch_size = size >> 2;
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(max_mask, insert_zero(min_mask, tid));
    hipFloatComplex tmp[4] = { {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.} };
    for (size_t p1 = 0; p1 < 2; p1++) {
      for (size_t p2 = 0; p2 < 2; p2++) {
        for (size_t q1 = 0; q1 < 2; q1++) {
          for (size_t q2 = 0; q2 < 2; q2++) {
            tmp[2 * q2 + q1] = hipCaddf(
              tmp[2 * q2 + q1],
              hipCmulf(
                q2g[8 * q2 + 4 * q1 + 2 * p2 + p1],
                state[stride2 * p2 + stride1 * p1 + btid]
              )
            );
          }
        }
      }
    }
    state[btid] = tmp[0];
    state[btid + stride1] = tmp[1];
    state[btid + stride2] = tmp[2];
    state[btid + stride1 + stride2] = tmp[3];
  }
}

extern "C"
int32_t q2gate(
  hipFloatComplex* state,
  const hipFloatComplex* gate,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  int32_t copy_status = hipMemcpyToSymbol(HIP_SYMBOL(q2g), gate, 16 * sizeof(hipFloatComplex));
  _q2gate<<<BLOCKS_NUM, THREADS_NUM>>>(state, pos2, pos1, qubits_number);
  return copy_status;
}

// one qubit density matrix computation
__global__ void _get_q1density(
  const hipFloatComplex* state,
  hipFloatComplex* density,
  size_t pos,
  size_t qubits_number
)
{
  __shared__ hipFloatComplex cache[4 * THREADS_NUM];
  size_t mask =  SIZE_MAX << pos;
  size_t stride = 1 << pos;
  size_t size = 1 << qubits_number;
  size_t batch_size = size >> 1;
  hipFloatComplex tmp[4] = { 
    {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(mask, tid);
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        tmp[2 * p + q] = hipCaddf(
          tmp[2 * p + q],
          hipCmulf(
            state[p * stride + btid],
            hipConjf(state[q * stride + btid])
          )
        );
      }
    }
  }
  for (int q = 0; q < 2; q++) {
    for (int p = 0; p < 2; p++) {
      cache[2 * p + q + 4 * threadIdx.x] = tmp[2 * p + q];
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q = 0; q < 2; q++) {
        for (int p = 0; p < 2; p++) {
          cache[2 * p + q + 4 * threadIdx.x] = hipCaddf(
            cache[2 * p + q + 4 * threadIdx.x],
            cache[2 * p + q + 4 * (threadIdx.x + s)]
          );
        }
      }
    }
    __syncthreads();
    s /= 2;
  }
  if (threadIdx.x == 0) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        density[2 * p + q + 4 * blockIdx.x] = cache[2 * p + q];
      }
    }
  }
}

extern "C"
int32_t get_q1density(
  const hipFloatComplex* state,
  hipFloatComplex* density,
  size_t pos,
  size_t qubits_number
)
{
  hipFloatComplex* device_density;
  hipFloatComplex* host_density;
  int32_t alloc_status = hipMalloc(&device_density, 4 * BLOCKS_NUM * sizeof(hipFloatComplex));
  _get_q1density<<<BLOCKS_NUM, THREADS_NUM>>>(
    state,
    device_density,
    pos,
    qubits_number
  );
  host_density = (hipFloatComplex*)malloc(4 * BLOCKS_NUM * sizeof(hipFloatComplex));
  int32_t memcopy_status = hipMemcpy(
    host_density,
    device_density,
    4 * BLOCKS_NUM * sizeof(hipFloatComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q = 0; q < 2; q++) {
      for (int p = 0; p < 2; p++) {
        density[2 * p + q] = hipCaddf(
          density[2 * p + q],
          host_density[2 * p + q + 4 * i]
        );
      }
    }
  }
  delete[] host_density;
  int32_t free_status = hipFree(device_density);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// two qubit density matrix computation
__global__ void _get_q2density(
  const hipFloatComplex* state,
  hipFloatComplex* density,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  __shared__ hipFloatComplex cache[16 * THREADS_NUM];
  size_t min_pos = MIN(pos1, pos2);
  size_t max_pos = MAX(pos1, pos2);
  size_t min_mask =  SIZE_MAX << min_pos;
  size_t max_mask =  SIZE_MAX << max_pos;
  size_t size = 1 << qubits_number;
  size_t stride1 = 1 << pos1;
  size_t stride2 = 1 << pos2;
  size_t batch_size = size >> 2;
  hipFloatComplex tmp[16] = {
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
    {0., 0.}, {0., 0.}, {0., 0.}, {0., 0.},
  };
  for (
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    tid < batch_size;
    tid += blockDim.x * gridDim.x
  )
  {
    size_t btid = insert_zero(max_mask, insert_zero(min_mask, tid));
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            tmp[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              tmp[8 * p2 + 4 * p1 + 2 * q2 + q1],
              hipCmulf(
                state[p2 * stride2 + p1 * stride1 + btid],
                hipConjf(state[q2 * stride2 + q1 * stride1 + btid])
              )
            );
          }
        }
      }
    }
  }
  for (int q1 = 0; q1 < 2; q1++) {
    for (int q2 = 0; q2 < 2; q2++) {
      for (int p1 = 0; p1 < 2; p1++) {
        for (int p2 = 0; p2 < 2; p2++) {
          cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = tmp[8 * p2 + 4 * p1 + 2 * q2 + q1];
        }
      }
    }
  }
  __syncthreads();
  int s = THREADS_NUM / 2;
  while ( s != 0 ) {
    if ( threadIdx.x < s ) {
      for (int q1 = 0; q1 < 2; q1++) {
        for (int q2 = 0; q2 < 2; q2++) {
          for (int p1 = 0; p1 < 2; p1++) {
            for (int p2 = 0; p2 < 2; p2++) {
              cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x] = hipCaddf(
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * threadIdx.x],
                cache[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * (threadIdx.x + s)]
              );
            }
          }
        }
      }
    }
    __syncthreads();
    s /= 2;
  }
  if (threadIdx.x == 0) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            density[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * blockIdx.x] = cache[8 * p2 + 4 * p1 + 2 * q2 + q1];
          }
        }
      }
    }
  }
}

extern "C"
int32_t get_q2density(
  const hipFloatComplex* state,
  hipFloatComplex* density,
  size_t pos2,
  size_t pos1,
  size_t qubits_number
)
{
  hipFloatComplex* device_density;
  hipFloatComplex* host_density;
  int32_t alloc_status = hipMalloc(&device_density, 16 * BLOCKS_NUM * sizeof(hipFloatComplex));
  _get_q2density<<<BLOCKS_NUM, THREADS_NUM>>>(
    state,
    device_density,
    pos2,
    pos1,
    qubits_number
  );
  host_density = (hipFloatComplex*)malloc(16 * BLOCKS_NUM * sizeof(hipFloatComplex));
  int32_t memcopy_status = hipMemcpy(
    host_density,
    device_density,
    16 * BLOCKS_NUM * sizeof(hipFloatComplex),
    hipMemcpyDeviceToHost
  );
  for (int i = 0; i < BLOCKS_NUM; i ++) {
    for (int q1 = 0; q1 < 2; q1++) {
      for (int q2 = 0; q2 < 2; q2++) {
        for (int p1 = 0; p1 < 2; p1++) {
          for (int p2 = 0; p2 < 2; p2++) {
            density[8 * p2 + 4 * p1 + 2 * q2 + q1] = hipCaddf(
              density[8 * p2 + 4 * p1 + 2 * q2 + q1],
              host_density[8 * p2 + 4 * p1 + 2 * q2 + q1 + 16 * i]
            );
          }
        }
      }
    }
  }
  delete[] host_density;
  int32_t free_status = hipFree(device_density);
  // return the first error code
  if ( alloc_status != 0 ) return alloc_status;
  if ( memcopy_status != 0 ) return memcopy_status;
  if ( free_status != 0 ) return free_status;
  return 0;
}

// copy of a state
__global__ void _copy(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  for (
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    dst[tid] = src[tid];
  }
}

extern "C"
void copy(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  _copy<<<BLOCKS_NUM, THREADS_NUM>>>(
    src,
    dst,
    qubits_number
  );
}

// primitives to pass gradient through the density matrix computation 
__global__ void _conj_and_double(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  for (
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    dst[tid].x = 2 * src[tid].x;
    dst[tid].y = -2 * src[tid].y;
  }
}

extern "C"
void conj_and_double(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  _conj_and_double<<<BLOCKS_NUM, THREADS_NUM>>>(
    src,
    dst,
    qubits_number
  );
}

__global__ void _add(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  for (
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    tid < (1 << qubits_number);
    tid += blockDim.x * gridDim.x
  )
  {
    dst[tid] = hipCaddf(dst[tid], src[tid]);
  }
}

extern "C"
void add(
  const hipFloatComplex* src,
  hipFloatComplex* dst,
  size_t qubits_number
)
{
  _add<<<BLOCKS_NUM, THREADS_NUM>>>(
    src,
    dst,
    qubits_number
  );
}
